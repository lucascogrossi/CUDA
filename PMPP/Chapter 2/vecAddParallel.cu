#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define N 1000

void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    float *A_d, *B_d, *C_d;
    int size = N * sizeof(float);

    hipMalloc((void**) &A_d, size);
    hipMalloc((void**) &B_d, size);
    hipMalloc((void**) &C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    vecAddkernel<<<ceil(n / 256.0), 256>>>(A_d, B_d, C_d, n);

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

__global__
void vecAddkernel(float* A, float* B, float* C, int n ) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

int main() {

}