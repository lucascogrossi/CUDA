
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>

inline hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

__global__ void matrixMultKernel(int* A, int* B, int* C, int N) {

    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;

    for (unsigned int i = 0; i < N; ++i) {
        sum += A[row * N + i] * B[i * N + col];
    }
    C[row * N + col] = sum;
}


void initMatrix(int* a, int N) {
    for(unsigned int i = 0; i < N * N; ++i)
        a[i] = rand() % 100;
}

void matrixMult(int* a_h, int* b_h, int* c_h, int N) {
    int *a_d, *b_d, *c_d;
    size_t size = sizeof(int) * N * N;

    checkCuda( hipMalloc((void**) &a_d, size) );
    checkCuda( hipMalloc((void**) &b_d, size) );
    checkCuda( hipMalloc((void**) &c_d, size ));

    checkCuda( hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice) );

    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((N + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x, 
                   (N + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);

    matrixMultKernel<<<numBlocks, numThreadsPerBlock>>>(a_d, b_d, c_d, N);
    hipDeviceSynchronize();

    checkCuda( hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost) );

    checkCuda( hipFree(a_d) );
    checkCuda( hipFree(b_d) );
    checkCuda( hipFree(c_d) );
}

int main() {

    int N = 1 << 10;
    size_t size = N * N * sizeof(int);

    int *a, *b, *c;

    a = (int*) malloc(size);
    b = (int*) malloc(size);
    c = (int*) malloc(size);

    initMatrix(a, N);
    initMatrix(b, N);

    matrixMult(a, b, c, N);

    free(a);
    free(b);
    free(c);
}