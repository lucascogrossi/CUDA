
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <cassert>

inline hipError_t checkCuda(hipError_t result) {
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

__global__ void matrixMultKernel(float* A, float* B, float* C, int N) {

    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    if ((row < N) && (col < N)) {
        float sum = 0.0f;

        for (unsigned int i = 0; i < N; ++i) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}


void initMatrix(float* a, int N) {
    for(unsigned int i = 0; i < N * N; ++i)
        a[i] = rand() % 100;
}

void matrixMult(float* a_h, float* b_h, float* c_h, int N) {
    float *a_d, *b_d, *c_d;
    size_t size = sizeof(float) * N * N;

    checkCuda( hipMalloc((void**) &a_d, size) );
    checkCuda( hipMalloc((void**) &b_d, size) );
    checkCuda( hipMalloc((void**) &c_d, size ));

    checkCuda( hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice) );

    dim3 numThreadsPerBlock(32, 32);
    dim3 numBlocks((N + numThreadsPerBlock.x - 1) / numThreadsPerBlock.x, 
                   (N + numThreadsPerBlock.y - 1) / numThreadsPerBlock.y);

    matrixMultKernel<<<numBlocks, numThreadsPerBlock>>>(a_d, b_d, c_d, N);
    hipDeviceSynchronize();

    checkCuda( hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost) );

    checkCuda( hipFree(a_d) );
    checkCuda( hipFree(b_d) );
    checkCuda( hipFree(c_d) );
}

int main() {

    int N = 1 << 10;
    size_t size = N * N * sizeof(float);

    float *a, *b, *c;

    a = (float*) malloc(size);
    b = (float*) malloc(size);
    c = (float*) malloc(size);

    initMatrix(a, N);
    initMatrix(b, N);

    matrixMult(a, b, c, N);

    free(a);
    free(b);
    free(c);
}