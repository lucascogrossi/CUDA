
#include <hip/hip_runtime.h>
int CHANNELS = 3; // We assume that CHANNELS is a constant of value 3, 
                  //and its definition is outside the kernel function.

// Each input image is encoded as unsigned chars [0, 255]
// Each pixel is 3 consecutive chars for the 3 channels (RGB)

__global__ void colorToGrayscaleConvertionKernel(unsigned char* grayImage, unsigned char* rgbImage, int width, int height) {
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        // Get 1D offset for the grayscale image
        int grayOffset = row * width + col;
        // One can think of the RGB image having CHANNEL
        // times more columns than the grayscale image
        int rgbOffset = grayOffset * CHANNELS;
        unsigned char r = rgbImage[rgbOffset    ]; // Red value
        unsigned char g = rgbImage[rgbOffset + 1]; // Green value
        unsigned char b = rgbImage[rgbOffset + 2]; // Blue value
        // Perform the rescaling and store it
        // We multiply by floating point constants
        grayImage[grayOffset] = 0.21f * r + 0.71f + 0.07f * b;
    }
}