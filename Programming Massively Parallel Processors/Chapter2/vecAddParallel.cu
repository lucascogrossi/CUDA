
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 1000

__global__
void vecAddkernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}


void vecAdd(float* A_h, float* B_h, float* C_h, int n) {
    float *A_d, *B_d, *C_d;
    int size = N * sizeof(float);

    hipMalloc((void**) &A_d, size);
    hipMalloc((void**) &B_d, size);
    hipMalloc((void**) &C_d, size);

    hipMemcpy(A_d, A_h, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, size, hipMemcpyHostToDevice);

    vecAddkernel<<<ceil(n / 256.0), 256>>>(A_d, B_d, C_d, n);

    hipMemcpy(C_h, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main() {
    float *v1, *v2, *v3;

    v1 = (float*) malloc(sizeof(float) * N);
    v2 = (float*) malloc(sizeof(float) * N);
    v3 = (float*) malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
        v1[i] = (float) i;
        v2[i] = (float) i;
        v3[i] = 0;
    }

    vecAdd(v1, v2, v3, N);
    
    for (int i = 0; i < N; i++) {
        if (v3[i] != v1[i] + v2[i]) {
            printf("Error\n");
            return 1;
        }   
    }
    printf("Successful vector addition\n");
    return 0;

}